
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void word_count(char *A, char *B, int *I, int *d_s, int spaces)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < spaces)
	{
		int flag = 1;
		int start = I[id] + 1;
		int end = I[id + 1];
		for(int i=0; i < end-start; i++)
			if(A[start + i] != B[i])
			{
				flag = 0;
				break;
			}
		if(flag == 1)
			atomicAdd(d_s, 1);
	}
}

int main()
{
	int i, size1, size2, size3, k = 0, spaces = 0;
	char *d_a, *d_b;
	int *d_i;
	int *search = 0, *d_s, *result;
	char *A = "PEOPLE NEED A NEW KING AND A NEW KING NEEDS A NEW CROWN";
	char *B = "NEW";
	int len = strlen(A);
	int len2 = strlen(B);

	for(i=0;i<len;i++)
		if(A[i] == ' ')
			spaces++;
	spaces++;

	int *I = (int *)calloc(spaces+1, sizeof(int));
	I[k++] = -1;

	for(i=0;i<len;i++)
		if(A[i] == ' ')
			I[k++] = i;
	I[k++] = len;

	size1 = len * sizeof(char);
	size2 = (spaces + 1) * sizeof(int);
	size3 = len2 * sizeof(char);

	hipMalloc((void **)&d_a, size1);
	hipMalloc((void **)&d_b, size3);
	hipMalloc((void **)&d_i, size2);
	hipMalloc((void **)&d_s, sizeof(unsigned int));

	hipMemcpy(d_a, A, size1, hipMemcpyHostToDevice);
	hipMemcpy(d_b, B, size3, hipMemcpyHostToDevice);
	hipMemcpy(d_i, I, size2, hipMemcpyHostToDevice);
	hipMemcpy(d_s, search, sizeof(unsigned int), hipMemcpyHostToDevice);

	word_count<<<ceil(spaces/256.0), 256>>>(d_a, d_b, d_i, d_s, spaces);

	hipMemcpy(result, d_s, sizeof(unsigned int), hipMemcpyDeviceToHost);

	printf("String : \n%s\n",A);
	printf("Total occurances of NEW :%d\n",result[0]);

	hipFree(d_a);
	hipFree(d_i);
	hipFree(d_s);
	hipFree(d_b);

	free(A);
	free(I);
	free(B);

	return 0;
}