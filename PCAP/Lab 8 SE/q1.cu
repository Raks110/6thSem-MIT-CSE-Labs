
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void str_repeat(char *A, char *B, int len, int N)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int i;
	if(id < N)
		for(i=0;i<len;i++)
			B[id*len + i] = A[i];
}

int main()
{
	char str[20];
	char *A, *B;
	int len, N, size1, size2;
	char *d_a, *d_b;

	printf("Enter a string : ");
	scanf("%s", str);

	len = strlen(str);
	A = (char *)malloc(sizeof(char) * len);
	strcpy(A, str);

	printf("Enter the repeat count : ");
	scanf("%d", &N);

	B = (char *)malloc(sizeof(char) * len * N);

	size1 = len * sizeof(char);
	size2 = N * len * sizeof(char);

	hipMalloc((void **)&d_a, size1);
	hipMalloc((void **)&d_b, size2);

	hipMemcpy(d_a, A, size1, hipMemcpyHostToDevice);

	str_repeat<<<ceil(N/256.0), 256>>>(d_a, d_b, len, N);

	hipMemcpy(B, d_b, size2, hipMemcpyDeviceToHost);

	printf("Output string :\n%s\n",B);

	hipFree(d_a);
	hipFree(d_b);

	free(A);
	free(B);

	return 0;
}