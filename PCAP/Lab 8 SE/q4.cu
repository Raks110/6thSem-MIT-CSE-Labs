
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void reverse_each_word(char *A, char *B, int len)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < len)
		B[id] = A[len - id - 1];
}

int main()
{
	int i, size;
	char *A = "GNINNIGEB WEN YM SI SIHT";

	int len = strlen(A);
	char *B = (char *)malloc(sizeof(char) * len);

	char *d_a, *d_b;

	size = len * sizeof(char);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);

	hipMemcpy(d_a, A, size, hipMemcpyHostToDevice);

	reverse_each_word<<<ceil(len/256.0), 256>>>(d_a, d_b, len);

	hipMemcpy(B, d_b, size, hipMemcpyDeviceToHost);

	printf("Original String : \n%s\n",A);
	printf("Desired Output :\n%s\n",B);

	hipFree(d_a);
	hipFree(d_b);

	free(B);

	return 0;
}