
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void reverse_each_word(char *A, int *I, char *B, int spaces)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < spaces)
	{
		int start = I[id] + 1;
		int end = I[id + 1];
		for(int i=0; i < (start + end)/2; i++)
		{
			B[end + start - 1] = A[i];
			B[i] = A[end + start - 1];
		}
	}
}

int main()
{
	int i, size1, size2, k = 0, spaces = 0;
	char *d_a, *d_b;
	int *d_i;
	char *A = "SIHT SI YM WEN GNINNIGEB";

	int len = strlen(A);
	char *B = (char *)malloc(sizeof(char) * len);

	for(i=0;i<len;i++)
		if(A[i] == ' ')
			spaces++;
	spaces++;

	int *I = (int *)malloc(sizeof(int) * (spaces+1));
	I[k++] = -1;

	for(i=0;i<len;i++)
		if(A[i] == ' ')
			I[k++] = i;
	I[k++] = len;

	size1 = len * sizeof(char);
	size2 = (spaces + 1) * sizeof(int);

	hipMalloc((void **)&d_a, size1);
	hipMalloc((void **)&d_i, size2);
	hipMalloc((void **)&d_b, size1);

	hipMemcpy(d_a, A, size1, hipMemcpyHostToDevice);
	hipMemcpy(d_i, I, size2, hipMemcpyHostToDevice);

	reverse_each_word<<<ceil(spaces/256.0), 256>>>(d_a, d_i, d_b, spaces);

	hipMemcpy(B, d_b, size1, hipMemcpyDeviceToHost);

	printf("Original String : \n%s\n",A);
	printf("Desired Output :\n%s\n",B);

	hipFree(d_a);
	hipFree(d_i);
	hipFree(d_b);

	free(B);
	free(I);

	return 0;
}