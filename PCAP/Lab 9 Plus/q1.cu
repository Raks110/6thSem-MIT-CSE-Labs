#include "hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>
#include<math.h>

__global__ void alter(int *a, int * t){
	int n = threadIdx.x, m=blockIdx.x, size = blockDim.x;
	t[m*size + n] = 1;
	int d = m;
	while(d>=0){
		t[m*size+n]*=a[m*size+n];
		d--;
	}
}

int main(void){
	int *a,*t,m,n,i,j;
	int *d_a,*d_t;

	printf("Enter the value of m : ");
	scanf("%d",&m);
	printf("Enter the value of n : ");
	scanf("%d",&n);

	int size = sizeof(int) * m * n;

	a = (int*)malloc(m*n*sizeof(int));
	t = (int*)malloc(m*n*sizeof(int));
	printf("Enter input matrix : \n");
	for(i=0;i<m*n;i++){
		scanf("%d",&a[i]);
	}
	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_t,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);

	alter<<<m,n>>>(d_a,d_t);

	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);

	printf("Result vector is : \n");
	for(i=0;i<m;i++){
		for(j=0;j<n;j++){
			printf("%d\t",t[i*n+j]);
		}
		printf("\n");
	}

	getchar();
	hipFree(d_a);
	hipFree(d_t);
	return 0;
}
