#include "hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>
#include<math.h>

__global__ void rowadd(int *a, int *b, int * t){
	int rows = blockDim.x, n=threadIdx.x;
	int c = gridDim.x; 
	for(int i=0;i<c;i++){
		t[n*c + i] = a[n*c + i] + b[n*c + i];
	}
}

__global__ void coladd(int *a, int *b, int * t){
	int col = blockDim.x, n=threadIdx.x;
	int r = gridDim.x;
	for(int i=0;i<r;i++){
		t[i*col + n] = a[i*col + n] + b[i*col + n];
	}
}

__global__ void eleadd(int *a, int *b, int * t){
	int n = threadIdx.x;
	t[n] = a[n]+b[n];
}

int main(void){
	int *a,*t,*b,m,n,i,j;
	int *d_a,*d_b,*d_t;

	printf("Enter the value of m : ");
	scanf("%d",&m);
	printf("Enter the value of n : ");
	scanf("%d",&n);

	int size = sizeof(int) * m * n;

	a = (int*)malloc(m*n*sizeof(int));
	b = (int*)malloc(m*n*sizeof(int));
	t = (int*)malloc(m*n*sizeof(int));

	printf("Enter input A matrix : \n");
	for(i=0;i<m*n;i++){
		scanf("%d",&a[i]);
	}

	printf("Enter input B matrix : \n");
	for(i=0;i<m*n;i++){
		scanf("%d",&b[i]);
	}

	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_b,size);
	hipMalloc((void**)&d_t,size);
	
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

	rowadd<<<n,m>>>(d_a,d_b,d_t);
	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
	printf("Result vector is : \n");
	for(i=0;i<m;i++){
		for(j=0;j<n;j++){
			printf("%d\t",t[i*n+j]);
		}
		printf("\n");
	}

	coladd<<<m,n>>>(d_a,d_b,d_t);
	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
	printf("Result vector is : \n");
	for(i=0;i<m;i++){
		for(j=0;j<n;j++){
			printf("%d\t",t[j*m+i]);
		}
		printf("\n");
	}

	eleadd<<<1,m*n>>>(d_a,d_b,d_t);
	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
	printf("Result vector is : \n");
	for(i=0;i<m;i++){
		for(j=0;j<n;j++){
			printf("%d\t",t[i*n+j]);
		}
		printf("\n");
	}

	getchar();
	hipFree(d_a);
	hipFree(d_t);
	return 0;
}
