
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void linear_op(int *a, int *b, int *c, int *scalar){

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	c[id] = (*scalar * a[id]) + b[id];
}

int main(){

	int N, alpha;

	printf("Enter ⍺: ");
	scanf("%d", &alpha);

	printf("Enter N: ");
	scanf("%d", &N);

	int vectorA[N], vectorB[N], vectorC[N];
	int *d_a, *d_b, *d_c, *d_d;

	hipMalloc((void**) &d_a, sizeof(int)*N);
	hipMalloc((void**) &d_b, sizeof(int)*N);
	hipMalloc((void**) &d_c, sizeof(int)*N);
	hipMalloc((void**) &d_d, sizeof(int));

	printf("Enter N Elements for A:\n");
	
	for(int i=0;i<N;i++){
		scanf("%d", &vectorA[i]);
	}

	printf("Enter N Elements for B:\n");
	for(int i=0;i<N;i++){
		scanf("%d", &vectorB[i]);
	}

	hipMemcpy(d_a, &vectorA, sizeof(int)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &vectorB, sizeof(int)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_d, &alpha, sizeof(int), hipMemcpyHostToDevice);

	linear_op<<<ceil(N/512.0), 512>>>(d_a, d_b, d_c, d_d);

	hipMemcpy(&vectorC, d_c, sizeof(int)*N, hipMemcpyDeviceToHost);
	
	for(int i=0;i<N;i++){
		printf("%d ", vectorC[i]);
	}
	
	printf("\n");

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}