
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void sine(float *a, float *b){

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	b[id] = sin(a[id]);
}

int main(){

	int N;

	printf("Enter N: ");
	scanf("%d", &N);

	float vectorA[N], vectorB[N];
	float *d_a, *d_b;

	hipMalloc((void**) &d_a, sizeof(float)*N);
	hipMalloc((void**) &d_b, sizeof(float)*N);

	printf("Enter N Angles (in Radians):\n");
	
	for(int i=0;i<N;i++){
		scanf("%f", &vectorA[i]);
	}

	hipMemcpy(d_a, &vectorA, sizeof(float)*N, hipMemcpyHostToDevice);

	sine<<<ceil(N/512.0), 512>>>(d_a, d_b);

	hipMemcpy(&vectorB, d_b, sizeof(float)*N, hipMemcpyDeviceToHost);
	
	for(int i=0;i<N;i++){
		printf("sin(%.3f) = %.3f\n", vectorA[i], vectorB[i]);
	}
	
	printf("\n");

	hipFree(d_a);
	hipFree(d_b);

	return 0;
}