
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void add_variable(int *a, int *b, int *c){

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	c[id] = a[id] + b[id];
}

__global__ void add_single_block(int *a, int *b, int *c){

	int id = threadIdx.x;
	c[id] = a[id] + b[id];
}

__global__ void add_single_thread(int *a, int *b, int *c){

	int id = blockDim.x * blockIdx.x;
	c[id] = a[id] + b[id];
}

int main(){

	int N;

	printf("Enter N: ");
	scanf("%d", &N);

	int vectorA[N], vectorB[N], vectorC[N];
	int *d_a, *d_b, *d_c;

	hipMalloc((void**) &d_a, sizeof(int)*N);
	hipMalloc((void**) &d_b, sizeof(int)*N);
	hipMalloc((void**) &d_c, sizeof(int)*N);

	printf("Enter N Elements for A:\n");
	for(int i=0;i<N;i++){
		scanf("%d", &vectorA[i]);
	}

	printf("Enter N Elements for B:\n");
	for(int i=0;i<N;i++){
		scanf("%d", &vectorB[i]);
	}

	hipMemcpy(d_a, &vectorA, sizeof(int)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &vectorB, sizeof(int)*N, hipMemcpyHostToDevice);

	printf("Choose what to do:\n1. Create 1 Thread each Block.\n2. Create 1 Block Only.\n3. Create DynamicTM.\nYour Choice: ");

	int choice = 1;

	scanf("%d", &choice);

	if(choice == 1)
		add_single_thread<<<N,1>>>(d_a, d_b, d_c);

	else if(choice == 2)
		add_single_block<<<1,N>>>(d_a, d_b, d_c);
	
	else
		add_variable<<<ceil(N/256.0),256>>>(d_a, d_b, d_c);

	hipMemcpy(&vectorC, d_c, sizeof(int)*N, hipMemcpyDeviceToHost);
	
	for(int i=0;i<N;i++){
		printf("%d ", vectorC[i]);
	}
	
	printf("\n");

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}